#include <iostream>
#include <hip/hip_runtime.h>

// Kernel CUDA para sumar vectores
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1 << 16; // 65536 elementos
    size_t size = N * sizeof(float);

    // Reservar memoria en host
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Inicializar vectores
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Reservar memoria en GPU
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copiar datos Host -> Device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Configurar grid y bloques
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Lanzar kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copiar resultado Device -> Host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verificar resultado
    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Liberar memoria
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
