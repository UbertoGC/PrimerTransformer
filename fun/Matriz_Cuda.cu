#include "hip/hip_runtime.h"
#include "Matriz2D.h"

#define BLOCK_SIZE 16

// ============================================
// Kernel: Multiplicación de matrices (float)
// ============================================
__global__ void matMulKernel(const float* A, const float* B, float* C,
                             int filasA, int colsA, int colsB) {
    __shared__ float tileA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tileB[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float val = 0.0f;

    for (int t = 0; t < (colsA + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        if (row < filasA && (t * BLOCK_SIZE + threadIdx.x) < colsA)
            tileA[threadIdx.y][threadIdx.x] = A[row * colsA + t * BLOCK_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < colsB && (t * BLOCK_SIZE + threadIdx.y) < colsA)
            tileB[threadIdx.y][threadIdx.x] = B[(t * BLOCK_SIZE + threadIdx.y) * colsB + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
            val += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];

        __syncthreads();
    }

    if (row < filasA && col < colsB)
        C[row * colsB + col] = val;
    
        
}

// ============================================
// Kernel: ReLU
// ============================================
__global__ void reluKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) data[idx] = fmaxf(0.0f, data[idx]);
}

// ============================================
// Kernel: Softmax por filas
// ============================================
__global__ void softmaxKernel(float* A, int rows, int cols) {
    extern __shared__ float shared[];
    int row = blockIdx.x;
    int tid = threadIdx.x;

    if (row >= rows) return;
    float* rowData = A + row * cols;

    // Max
    float maxVal = -1e30f;
    for (int j = tid; j < cols; j += blockDim.x)
        maxVal = fmaxf(maxVal, rowData[j]);
    shared[tid] = maxVal;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) shared[tid] = fmaxf(shared[tid], shared[tid + stride]);
        __syncthreads();
    }
    maxVal = shared[0];

    // Exp y suma
    float sum = 0.0f;
    for (int j = tid; j < cols; j += blockDim.x) {
        rowData[j] = expf(rowData[j] - maxVal);
        sum += rowData[j];
    }
    shared[tid] = sum;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) shared[tid] += shared[tid + stride];
        __syncthreads();
    }
    sum = shared[0];

    for (int j = tid; j < cols; j += blockDim.x)
        rowData[j] /= sum;
}

// ============================================
// Kernel: Normalización por filas
// ============================================
__global__ void normalizeKernel(float* A, int rows, int cols) {
    int row = blockIdx.x;
    if (row >= rows) return;

    extern __shared__ float shared[];
    float* mean = shared;
    float* var = shared + 1;

    if (threadIdx.x == 0) {
        *mean = 0.0f;
        *var = 0.0f;
    }
    __syncthreads();

    atomicAdd(mean, A[row * cols + threadIdx.x]);
    __syncthreads();

    if (threadIdx.x == 0) *mean /= cols;
    __syncthreads();

    float diff = A[row * cols + threadIdx.x] - *mean;
    atomicAdd(var, diff * diff);
    __syncthreads();

    if (threadIdx.x == 0) *var = sqrtf(*var / cols);
    __syncthreads();

    if (*var > 0)
        A[row * cols + threadIdx.x] = diff / *var;
}

// ============================================
// Wrappers CUDA en Matriz2D
// ============================================
void Matriz2D::RELU_CUDA() {
    size_t size = filas * columnas;
    float *d_data;
    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    reluKernel<<<blocks, threads>>>(d_data, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_data));
}

void Matriz2D::SoftmaxFilas_CUDA() {
    size_t size = filas * columnas;
    float *d_data;
    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    size_t shared_mem = threads * sizeof(float);
    softmaxKernel<<<filas, threads, shared_mem>>>(d_data, filas, columnas);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_data));
}

Matriz2D Matriz2D::MultiplicarCUDA(const Matriz2D& B) const {
    if (columnas != B.filas)
        throw std::invalid_argument("Dimensiones incompatibles para multiplicación");

    Matriz2D R(filas, B.columnas);

    float *d_A, *d_B, *d_C;
    size_t sizeA = filas * columnas * sizeof(float);
    size_t sizeB = B.filas * B.columnas * sizeof(float);
    size_t sizeC = filas * B.columnas * sizeof(float);

    CUDA_CHECK(hipMalloc(&d_A, sizeA));
    CUDA_CHECK(hipMalloc(&d_B, sizeB));
    CUDA_CHECK(hipMalloc(&d_C, sizeC));

    CUDA_CHECK(hipMemcpy(d_A, datos, sizeA, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B.datos, sizeB, hipMemcpyHostToDevice));

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((B.columnas + BLOCK_SIZE - 1) / BLOCK_SIZE,
                (filas + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, filas, columnas, B.columnas);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(R.datos, d_C, sizeC, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    return R;
}
__global__ void normalizeKernelGammaBeta(float* A, const float* gamma, const float* beta,
                                          int rows, int cols, float epsilon) {
    int row = blockIdx.x;
    if (row >= rows) return;

    extern __shared__ float shared[];
    float* sum = shared;
    float* sumSq = shared + 1;

    if (threadIdx.x == 0) {
        *sum = 0.0f;
        *sumSq = 0.0f;
    }
    __syncthreads();

    // Calcular suma y suma de cuadrados
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
        float val = A[row * cols + j];
        atomicAdd(sum, val);
        atomicAdd(sumSq, val * val);
    }
    __syncthreads();

    float mean = *sum / cols;
    float var = (*sumSq / cols) - (mean * mean);
    float invStd = rsqrtf(var + epsilon);

    // Normalización + γ y β
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
        int idx = row * cols + j;
        float normVal = (A[idx] - mean) * invStd;
        A[idx] = normVal * gamma[j] + beta[j];
    }
}

void Matriz2D::NormalizarFilas_CUDA(const Matriz2D& gamma, const Matriz2D& beta) {
    if (gamma.fil() != 1 || beta.fil() != 1 || gamma.col() != columnas || beta.col() != columnas) {
        throw std::runtime_error("Dimensiones de gamma/beta no compatibles con NormalizarFilas_CUDA");
    }

    int size = filas * columnas;
    float *d_data, *d_gamma, *d_beta;

    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_gamma, columnas * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_beta, columnas * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_gamma, gamma.Datos(), columnas * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_beta, beta.Datos(), columnas * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    size_t shared_mem = 2 * sizeof(float); // sum y sumSq
    normalizeKernelGammaBeta<<<filas, threads, shared_mem>>>(d_data, d_gamma, d_beta, filas, columnas, 1e-6f);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_data);
    hipFree(d_gamma);
    hipFree(d_beta);
}


__global__ void KernelSumarFila(float* datos, const float* bias, int filas, int columnas) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < filas * columnas) {
        int col = idx % columnas;
        datos[idx] += bias[col];
    }
}
__global__ void KernelSumarMatrices(float* A, const float* B, int total) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        A[idx] += B[idx];
    }
}


void Matriz2D::SumarFilaCUDA(const Matriz2D& fila) {
    if (fila.fil() != 1 && fila.fil() != filas) {
        std::cerr << "Error: La matriz no es compatible para broadcast en GPU." << std::endl;
        return;
    }

    int size = filas * columnas;
    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    float* d_datos;
    float* d_bias;
    CUDA_CHECK(hipMalloc(&d_datos, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_bias, columnas * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_datos, datos, size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_bias, fila.Datos(), columnas * sizeof(float), hipMemcpyHostToDevice));

    KernelSumarFila<<<blocks, threads>>>(d_datos, d_bias, filas, columnas);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_datos, size * sizeof(float), hipMemcpyDeviceToHost));
    hipFree(d_datos);
    hipFree(d_bias);
}
void Matriz2D::SumarMatrizCUDA(const Matriz2D& otra) {
    if (filas != otra.Filas() || columnas != otra.Columnas()) {
        throw std::runtime_error("Dimensiones incompatibles en SumarMatrizCUDA");
    }

    int total = filas * columnas;
    float *d_A, *d_B;

    CUDA_CHECK(hipMalloc(&d_A, total * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, total * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, datos, total * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, otra.Datos(), total * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (total + threads - 1) / threads;

    KernelSumarMatrices<<<blocks, threads>>>(d_A, d_B, total);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_A, total * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_A);
    hipFree(d_B);
}

__global__ void KernelEscalar(float* datos, float escalar, int total) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        datos[idx] *= escalar;
    }
}

void Matriz2D::EscalarCUDA(float escalar) {
    int total = filas * columnas;
    float* d_datos;
    hipMalloc(&d_datos, total * sizeof(float));
    hipMemcpy(d_datos, datos, total * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    KernelEscalar<<<blocks, threads>>>(d_datos, escalar, total);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    hipDeviceSynchronize();

    hipMemcpy(datos, d_datos, total * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_datos);
}
